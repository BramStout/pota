#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <map>
#include "../../Eigen/Eigen/Core"
#include "../../Eigen/Eigen/LU"
#include <fstream>


#define TINYEXR_IMPLEMENTATION
#include "tinyexr.h"


#include <chrono>


std::string replace_first_occurence(std::string& s, const std::string& toReplace, const std::string& replaceWith) {
    std::size_t pos = s.find(toReplace);
    if (pos == std::string::npos) return s;
    return s.replace(pos, toReplace.length(), replaceWith);
}

void save_to_exr_rgba(std::vector<float> img, std::string filename, unsigned xres, unsigned yres) {
  EXRHeader header;
  InitEXRHeader(&header);

  EXRImage image;
  InitEXRImage(&image);
  image.num_channels = 4;
  image.width = xres;
  image.height = yres;

  std::vector<float> images[4];
  images[0].resize(xres * yres);
  images[1].resize(xres * yres);
  images[2].resize(xres * yres);
  images[3].resize(xres * yres);

  for (unsigned int i = 0; i < xres * yres; i++) {
    images[0][i] = img[4*i+0];
    images[1][i] = img[4*i+1];
    images[2][i] = img[4*i+2];
    images[3][i] = img[4*i+3];
  }

  float* image_ptr[4];
  image_ptr[0] = &(images[3].at(0)); // A
  image_ptr[1] = &(images[2].at(0)); // B
  image_ptr[2] = &(images[1].at(0)); // G
  image_ptr[3] = &(images[0].at(0)); // R

  image.images = (unsigned char**)image_ptr;
  header.num_channels = 4;
  header.channels = (EXRChannelInfo *)malloc(sizeof(EXRChannelInfo) * header.num_channels);
  strncpy(header.channels[0].name, "A", 255); header.channels[0].name[strlen("A")] = '\0';
  strncpy(header.channels[1].name, "B", 255); header.channels[1].name[strlen("B")] = '\0';
  strncpy(header.channels[2].name, "G", 255); header.channels[2].name[strlen("G")] = '\0';
  strncpy(header.channels[3].name, "R", 255); header.channels[3].name[strlen("R")] = '\0';

  header.pixel_types = (int *)malloc(sizeof(int) * header.num_channels); 
  header.requested_pixel_types = (int *)malloc(sizeof(int) * header.num_channels);
  for (int i = 0; i < header.num_channels; i++) {
    header.pixel_types[i] = TINYEXR_PIXELTYPE_FLOAT; // pixel type of input image
    header.requested_pixel_types[i] = TINYEXR_PIXELTYPE_HALF; // pixel type of output image to be stored in .EXR
  }
  
  header.compression_type = TINYEXR_COMPRESSIONTYPE_ZIP;

  const char* err;
  int ret = SaveEXRImageToFile(&image, &header, filename.c_str(), &err);
  if (ret != TINYEXR_SUCCESS) {
    std::cout << "[LENTIL BIDIRECTIONAL TL] Error when saving exr: " << err << std::endl;
  }
}


// xorshift fast random number generator
__device__ uint32_t xor128(void){
  static uint32_t x = 123456789, y = 362436069, z = 521288629, w = 88675123;
  uint32_t t = x ^ (x << 11);
  x = y; y = z; z = w;
  return w = (w ^ (w >> 19) ^ t ^ (t >> 8));
}


// sin approximation, not completely accurate but faster than std::sin
__device__ float fast_sin(float x){
    x = fmod(x + M_PI, M_PI * 2) - M_PI; // restrict x so that -M_PI < x < M_PI
    const float B = 4.0f / M_PI;
    const float C = -4.0f / (M_PI*M_PI);
    float y = B * x + C * x * std::abs(x);
    const float P = 0.225f;
    return P * (y * std::abs(y) - y) + y;
}


__device__ float fast_cos(float x){
    // conversion from sin to cos
    x += M_PI * 0.5;

    x = fmod(x + M_PI, M_PI * 2) - M_PI; // restrict x so that -M_PI < x < M_PI
    const float B = 4.0f / M_PI;
    const float C = -4.0f / (M_PI*M_PI);
    float y = B * x + C * x * std::abs(x);
    const float P = 0.225f;
    return P * (y * std::abs(y) - y) + y;
}


// Improved concentric mapping code by Dave Cline [peter shirley´s blog]
// maps points on the unit square onto the unit disk uniformly
__device__ void concentricDiskSample(float ox, float oy, Eigen::Vector2d &lens) {
    if (ox == 0.0 && oy == 0.0){
        lens(0) = 0.0;
        lens(1) = 0.0;
        return;
    }

    float phi, r;

    // switch coordinate space from [0, 1] to [-1, 1]
    const float a = 2.0 * ox - 1.0;
    const float b = 2.0 * oy - 1.0;

    if ((a * a) > (b * b)){
        r = a;
        phi = 0.78539816339 * (b / a);
    }
    else {
        r = b;
        phi = (1.57079632679) - ((0.78539816339) * (a / b));
    }


    bool fast_trigo = false;

    const float cos_phi = cosf(phi);
    const float sin_phi =  sinf(phi);
    lens(0) = r * cos_phi;
    lens(1) = r * sin_phi;
}


inline float clamp_min(float in, const float min) {
    if (in < min) in = min;
    return in;
}

inline float clamp(float in, const float min, const float max) {
    if (in < min) in = min;
    if (in > max) in = max;
    return in;
}

__device__ float thinlens_get_image_dist_focusdist(const float focal_length, const float focus_distance){
    return (-focal_length * -focus_distance) / (-focal_length + -focus_distance);
}


 __device__ float rand(Eigen::Vector2f vec){
    Eigen::Vector2f constant(12.9898,78.233);
    float dotproduct = vec.dot(constant);
    float tmp = sinf(dotproduct) * 43758.5453;
    return tmp - floor(tmp);
}


__global__ void trace_backwards(Eigen::Vector4d *image, Eigen::Vector4d *image_unredist, Eigen::Vector4d *image_redist,
                                      float *redist_weight_per_pixel, float *unredist_weight_per_pixel, float *zbuffer,
                                      const Eigen::Vector3d *sample_pos_cs, const float *focal_length, const float *aperture_radius, 
                                      const float *focus_distance, const float *sensor_width, const float *frame_aspect_ratio,
                                      const int *xres, const int *yres,
                                      const Eigen::Vector4d *sample, const int *samples, const float *inv_density) {

  
  const Eigen::Vector3d camera_space_sample_position_mb = *sample_pos_cs;
  const float image_dist_samplepos_mb = (-*focal_length * camera_space_sample_position_mb(2)) / (-*focal_length + camera_space_sample_position_mb(2));



  int i = threadIdx.x + blockIdx.x * blockDim.x;
  Eigen::Vector2f randseed1(float(i), float(i+123));
  int i2 = threadIdx.x + blockIdx.x * blockDim.x + 12345;
  Eigen::Vector2f randseed2(float(i2), float(i2+123));
  float r1 = rand(randseed1);
  float r2 = rand(randseed2);
  



  // either get uniformly distributed points on the unit disk or bokeh image
  Eigen::Vector2d unit_disk(0, 0);
  // float r1 = xor128() / 4294967296.0;
  // float r2 = xor128() / 4294967296.0;
  // printf("r1: %f, r2: %f", r1, r2);
  concentricDiskSample(r1, r2, unit_disk);
  
  // ray through center of lens
  Eigen::Vector3d dir_tobase = camera_space_sample_position_mb.normalized();
  float samplepos_image_intersection = std::abs(image_dist_samplepos_mb/dir_tobase(2));
  Eigen::Vector3d samplepos_image_point = dir_tobase * samplepos_image_intersection;

  // depth of field
  Eigen::Vector3d lens(unit_disk(0) * *aperture_radius, unit_disk(1) * *aperture_radius, 0.0);
  Eigen::Vector3d dir_from_lens_to_image_sample = samplepos_image_point - lens;
  dir_from_lens_to_image_sample.normalize();
  float focusdist_intersection = std::abs(thinlens_get_image_dist_focusdist(*focal_length, *focus_distance)/dir_from_lens_to_image_sample(2));
  

  Eigen::Vector3d focusdist_image_point = lens + dir_from_lens_to_image_sample*focusdist_intersection;
  
  // takes care of correct screenspace coordinate mapping
  Eigen::Vector2d sensor_position(focusdist_image_point(0) / focusdist_image_point(2),
                                  focusdist_image_point(1) / focusdist_image_point(2));
  sensor_position /= (*sensor_width*0.5)/-*focal_length;


  // optical vignetting
  Eigen::Vector3d dir_lens_to_P = camera_space_sample_position_mb - lens;
  dir_lens_to_P.normalize();


  // convert sensor position to pixel position
  float frame_aspect_ratio_tmp = *frame_aspect_ratio;
  // printf("frame aspect: %f", *frame_aspect_ratio);
  const float pixel_x = (( sensor_position(0) + 1.0) / 2.0) * *xres;
  const float pixel_y = ((-sensor_position(1) * frame_aspect_ratio_tmp + 1.0) / 2.0) * *yres;
  // printf("%f %f \n", pixel_x, pixel_y);

  // if outside of image
  if ((pixel_x >= *xres) || (pixel_x < 0) || (pixel_y >= *yres) || (pixel_y < 0)) return;

  // write sample to image
  unsigned pixelnumber = static_cast<int>(*xres * floor(pixel_y) + floor(pixel_x));

  
  Eigen::Vector4d rgba_energy = *sample / (double)(*samples);
  image_redist[pixelnumber] += rgba_energy * *inv_density;
  redist_weight_per_pixel[pixelnumber] += *inv_density / double(*samples);
  

  return;
}


int main() {
  
  // read the sampledata into vectors
  std::ifstream infile("/home/cactus/lentil/pota/tests/cuda/sampledata.txt");
  float sample_r, sample_g, sample_b, sample_a, depth, sample_pos_ws_x, sample_pos_ws_y, sample_pos_ws_z;
  std::vector<Eigen::Vector4d> sample_list;
  std::vector<Eigen::Vector3d> pos_ws_list;
  std::vector<float> depth_list;
  while (infile >> sample_r >> sample_g >> sample_b >> sample_a >> depth >> sample_pos_ws_x >> sample_pos_ws_y >> sample_pos_ws_z)
  {
      sample_list.push_back(Eigen::Vector4d(sample_r, sample_g, sample_b, sample_a));
      depth_list.push_back(depth);
      pos_ws_list.push_back(Eigen::Vector3d(sample_pos_ws_x, sample_pos_ws_y, sample_pos_ws_z));
  }



  int xres = 1920;
  int yres = 1080;
  int framenumber = 1;
  int aa_samples = 4;
  
  
  Eigen::Vector4d *image_device, *image_unredist_device, *image_redist_device;
  hipMalloc((void **)&image_device, xres*yres*sizeof(Eigen::Vector4d));
  hipMalloc((void **)&image_unredist_device, xres*yres*sizeof(Eigen::Vector4d));
  hipMalloc((void **)&image_redist_device, xres*yres*sizeof(Eigen::Vector4d));

  float *redist_weight_per_pixel_device, *unredist_weight_per_pixel_device, *zbuffer_device;
  hipMalloc((void **)&redist_weight_per_pixel_device, xres*yres*sizeof(float));
  hipMalloc((void **)&unredist_weight_per_pixel_device, xres*yres*sizeof(float));
  hipMalloc((void **)&zbuffer_device, xres*yres*sizeof(float));




  float sensor_width;
  float focal_length;
  float fstop;
  float focus_distance;
  float aperture_radius;
  float abb_spherical;
  float circle_to_square;
  float bokeh_anamorphic;
  std::string bidir_output_path;
  unsigned int bidir_sample_mult;

  sensor_width = 36.0;
  focal_length = 50.0;
  focal_length = clamp_min(focal_length, 0.01);
  fstop = 1.4;
  fstop = clamp_min(fstop, 0.01);
  focus_distance = 35.0;
  aperture_radius = (focal_length / (2.0 * fstop)) / 10.0;
  bidir_output_path = "/home/cactus/lentil/pota/tests/cuda/cuda.tl.<aov>.<frame>.exr";
  abb_spherical = 0.5;
  abb_spherical = clamp(abb_spherical, 0.001, 0.999);
  circle_to_square = 0.0;
  circle_to_square = clamp(circle_to_square, 0.01, 0.99);
  bokeh_anamorphic = 1.0;
  bokeh_anamorphic = clamp(bokeh_anamorphic, 0.01, 99999.0);
  bidir_sample_mult = 10;


  

  std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();




  const float d_xres = (float)xres;
  const float d_yres = (float)yres;
  const float frame_aspect_ratio = d_xres/d_yres;
  std::cout << "frame aspect ratio: " << frame_aspect_ratio << std::endl;




  int *xres_device, *yres_device, *samples_device;
  float *focal_length_device, *aperture_radius_device, *focus_distance_device, *sensor_width_device, *frame_aspect_ratio_device;

  hipMalloc((void **)&xres_device, sizeof(int));
  hipMalloc((void **)&yres_device, sizeof(int));
  hipMalloc((void **)&samples_device, sizeof(int));

  hipMalloc((void **)&focal_length_device, sizeof(float));
  hipMalloc((void **)&aperture_radius_device, sizeof(float));
  hipMalloc((void **)&focus_distance_device, sizeof(float));
  hipMalloc((void **)&sensor_width_device, sizeof(float));
  hipMalloc((void **)&frame_aspect_ratio_device, sizeof(float));

  hipMemcpy(focal_length_device, &focal_length, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(aperture_radius_device, &aperture_radius, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(focus_distance_device, &focus_distance, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(sensor_width_device, &sensor_width, sizeof(float), hipMemcpyHostToDevice);	
  hipMemcpy(frame_aspect_ratio_device, &frame_aspect_ratio, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(xres_device, &xres, sizeof(int), hipMemcpyHostToDevice);	
  hipMemcpy(yres_device, &yres, sizeof(int), hipMemcpyHostToDevice);


  for (int i=0; i<sample_list.size(); ++i) {

    // std::cout << "count: " << i << std::endl;


    Eigen::Vector4d sample = sample_list[i];
    
    const Eigen::Vector3d sample_pos_cs = pos_ws_list[i];
    float depth = depth_list[i];
    const float inv_density = 1.0/16.0;
    
    const Eigen::Vector4d sample_transmission = Eigen::Vector4d(0,0,0,0);
    bool transmitted_energy_in_sample = ((sample_transmission(0)) > 0.0);
    if (transmitted_energy_in_sample){
      sample(0) -= sample_transmission(0);
      sample(1) -= sample_transmission(1);
      sample(2) -= sample_transmission(2);
    }




      // Eigen::Matrix4d world_to_camera_matrix;
      // world_to_camera_matrix << 1.0, 0.0, 0.0, 0.0,
      //                           0.0, 1.0, 0.0, 0.0,
      //                           0.0, 0.0, 1.0, -75.0,
      //                           0.0, 0.0, 0.0, 1.0;

      const Eigen::Vector3d camera_space_sample_position_static = sample_pos_cs;
      
      const float image_dist_samplepos = (-focal_length * camera_space_sample_position_static(2)) / (-focal_length + camera_space_sample_position_static(2));
      const float image_dist_focusdist = (-focal_length * -focus_distance) / (-focal_length + -focus_distance);
      float circle_of_confusion = std::abs((aperture_radius * (image_dist_samplepos - image_dist_focusdist))/image_dist_samplepos); // coc diameter
      


      const float coc_squared_pixels = std::pow(circle_of_confusion * yres, 2) * bidir_sample_mult * 0.01; // pixel area as baseline for sample count
      // if (std::pow(circle_of_confusion * yres, 2) < std::pow(15, 2)) goto no_redist; // 15^2 px minimum coc
      int samples = std::ceil(coc_squared_pixels / (double)std::pow(aa_samples, 2)); // aa_sample independence
      samples = clamp(samples, 100, 1000000);


      
      

// TO-PARALLELIZE
      // unsigned total_samples_taken = 0;
      // unsigned count = 0;
      // while(count<samples && ++total_samples_taken < samples*10) {


    float *inv_density_device;
    Eigen::Vector4d *sample_device;
    Eigen::Vector3d *sample_pos_cs_device;
    hipMalloc((void **)&inv_density_device, sizeof(float));
    hipMalloc((void **)&sample_device, sizeof(Eigen::Vector4d));
    hipMalloc((void **)&sample_pos_cs_device, sizeof(Eigen::Vector3d));

    hipMemcpy(inv_density_device, &inv_density, sizeof(float), hipMemcpyHostToDevice);	
    hipMemcpy(sample_device, &sample, sizeof(Eigen::Vector4d), hipMemcpyHostToDevice);	
    hipMemcpy(sample_pos_cs_device, &sample_pos_cs, sizeof(Eigen::Vector3d), hipMemcpyHostToDevice);
      
      
    trace_backwards<<<4, 512>>>(image_device, image_unredist_device, image_redist_device,
                    redist_weight_per_pixel_device, unredist_weight_per_pixel_device, zbuffer_device,
                    sample_pos_cs_device, focal_length_device, aperture_radius_device, 
                    focus_distance_device, sensor_width_device, frame_aspect_ratio_device,
                    xres_device, yres_device,
                    sample_device, samples_device, inv_density_device);
        
        // if (!success) continue;
        // ++count;
      // }
      
    

      
    hipFree(inv_density_device);
    hipFree(samples_device);
    hipFree(sample_device);
    hipFree(sample_pos_cs_device);
  }

  hipDeviceSynchronize();

  hipFree(xres_device);
  hipFree(yres_device);
  hipFree(focal_length_device);
  hipFree(aperture_radius_device);
  hipFree(focus_distance_device);
  hipFree(sensor_width_device);
  hipFree(frame_aspect_ratio_device);
  
    
  Eigen::Vector4d *image = new Eigen::Vector4d[xres*yres];
  Eigen::Vector4d *image_unredist = new Eigen::Vector4d[xres*yres];
  Eigen::Vector4d *image_redist = new Eigen::Vector4d[xres*yres];
  float *redist_weight_per_pixel = new float[xres*yres];
  float *unredist_weight_per_pixel = new float[xres*yres];
  float *zbuffer = new float[xres*yres];



  // hipMemcpy(&image, image_device, xres*yres*sizeof(Eigen::Vector4d), hipMemcpyDeviceToHost);
  // hipMemcpy(&image_unredist, image_unredist_device, xres*yres*sizeof(Eigen::Vector4d), hipMemcpyDeviceToHost);
  // hipMemcpy(&image_redist, image_redist_device, xres*yres*sizeof(Eigen::Vector4d), hipMemcpyDeviceToHost);
  // hipMemcpy(&redist_weight_per_pixel, redist_weight_per_pixel_device, xres*yres*sizeof(float), hipMemcpyDeviceToHost);
  // hipMemcpy(&unredist_weight_per_pixel, unredist_weight_per_pixel_device, xres*yres*sizeof(float), hipMemcpyDeviceToHost);
  // hipMemcpy(&zbuffer, zbuffer_device, xres*yres*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(image, image_device, xres*yres*sizeof(Eigen::Vector4d), hipMemcpyDeviceToHost);
  hipMemcpy(image_unredist, image_unredist_device, xres*yres*sizeof(Eigen::Vector4d), hipMemcpyDeviceToHost);
  hipMemcpy(image_redist, image_redist_device, xres*yres*sizeof(Eigen::Vector4d), hipMemcpyDeviceToHost);
  hipMemcpy(redist_weight_per_pixel, redist_weight_per_pixel_device, xres*yres*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(unredist_weight_per_pixel, unredist_weight_per_pixel_device, xres*yres*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(zbuffer, zbuffer_device, xres*yres*sizeof(float), hipMemcpyDeviceToHost);

  
  std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
  std::cout << "Time difference (sec) = " << (std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()) /1000000.0 <<std::endl;
  





// CLOSE


  std::vector<float> imageexr(yres * xres * 4);
  int offset = -1;

  for(unsigned px = 0; px < xres * yres; px++){

    Eigen::Vector4d redist = image_redist[px] / ((redist_weight_per_pixel[px] == 0.0) ? 1.0 : redist_weight_per_pixel[px]);
    Eigen::Vector4d unredist = image_unredist[px] / ((unredist_weight_per_pixel[px] == 0.0) ? 1.0 : unredist_weight_per_pixel[px]);
    Eigen::Vector4d combined_redist_unredist = (unredist * (1.0-redist_weight_per_pixel[px])) + (redist * (redist_weight_per_pixel[px]));
    // if (image_redist[px](0) > 0.0) std::cout << image_redist[px] << std::endl;
    if (combined_redist_unredist(3) > 0.95) combined_redist_unredist /= combined_redist_unredist(3);

    imageexr[++offset] = combined_redist_unredist(0);
    imageexr[++offset] = combined_redist_unredist(1);
    imageexr[++offset] = combined_redist_unredist(2);
    imageexr[++offset] = combined_redist_unredist(3);
  
  }

  // replace <aov> and <frame>
  std::string path = bidir_output_path;
  std::string path_replaced_aov = replace_first_occurence(path, "<aov>", "RGBA");
  
  std::string frame_str = std::to_string(framenumber);
  std::string frame_padded = std::string(4 - frame_str.length(), '0') + frame_str;
  std::string path_replaced_framenumber = replace_first_occurence(path, "<frame>", frame_padded);

  // dump framebuffers to exrs
  save_to_exr_rgba(imageexr, path_replaced_framenumber, xres, yres);

  printf("written image!");

  hipFree(image_device);
  hipFree(image_unredist_device);
  hipFree(image_redist_device);
  hipFree(redist_weight_per_pixel_device);
  hipFree(unredist_weight_per_pixel_device);
  hipFree(zbuffer_device);


  return 0;
}